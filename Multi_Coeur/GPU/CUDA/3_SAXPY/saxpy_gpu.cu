#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define THREAD_PER_BLOCK 256

/********************** kernel **************************/
__global__ void saxpy(int n, float a, float *x, float *y) {
    /* TODO : Calcul de l'indice i */
    //int i_vect_line = blockIdx.x;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    /* Calcul de saxpy */
    if (i < n) 
        y[i] = a * x[i] + y[i];
}

/********************** main **************************/
int main(void) {
    int N = 1 << 20;
    unsigned long vec_size = sizeof(float) * N;
    float *x, *y, *gpu_x, *gpu_y;
    
    x = (float*) malloc(vec_size);
    y = (float*) malloc(vec_size);

    /* Allocation de l'espace pour gpu_x et gpu_y qui vont recevoir x et y sur le GPU */
    hipMalloc((void **) &gpu_x, vec_size);
    hipMalloc((void **) &gpu_y, vec_size);
    
    /* Initialisation de x et y */
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    /* Copie de x et y sur le GPU dans gpu_x et gpu_y respectivement */
    hipMemcpy(gpu_x, x, vec_size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_y, y, vec_size, hipMemcpyHostToDevice);
    
    /* Appel au kernel saxpy sur les N éléments avec a = 2.0f */
    saxpy<<<N/THREAD_PER_BLOCK, THREAD_PER_BLOCK>>>(N, 2.0, gpu_x, gpu_y);
    
    /* Copie du résultat dans y */
    hipMemcpy(y, gpu_y, vec_size, hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        if(y[i] != 4.0f)
            printf("not equal %d %f %f\n", i, y[i], x[i]);

        maxError = max(maxError, abs(y[i]-4.0f));
    }
    printf("Max error: %f\n", maxError);

    /* Libération de la mémoire sur le GPU */
    hipFree(gpu_x);
    hipFree(gpu_x);

    free(x);
    free(y);
}
