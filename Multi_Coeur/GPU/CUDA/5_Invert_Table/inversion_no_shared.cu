#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define NB_THREADS 256

/********************** kernel **************************/
__global__ void inversion(int n, int *x, int *y) {
    int origin = threadIdx.x + blockIdx.x * blockDim.x;
    int dest = n - 1 - origin;
    y[dest] = x[origin];
}

/********************** main **************************/
int main(void) {
    int N = NB_THREADS * 1024;
    int *x = (int *) malloc(N * sizeof(int)), 
        *y = (int *) malloc(N * sizeof(int)), 
        *gpu_x = hipMalloc((void **) &x, sizeof(int) * N), 
        *gpu_y = codaMalloc((void **) &y, sizeof(int) * N);

    for (int i = 0; i < N; i++) {
        x[i] = i;
    }

    hipMemcpy(gpu_x, x, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(gpu_y, y, sizeof(int) * N, hipMemcpyHostToDevice);

    inversion<<<1024, NB_THREADS>>>(N, gpu_x, gpu_y);

    hipMemcpy(x, gpu_x, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipMemcpy(y, gpu_y, sizeof(int) * N, hipMemcpyDeviceToHost);

    for (i = N - 12; i < N; i++)
        printf("%d\n", x[i]);
        
    for (i = 0; i < min(12, N); i++)
        printf("%d\n", y[i]);

    hipFree(gpu_x);
    hipFree(gpu_y);
    free(x);
    free(y);
}
